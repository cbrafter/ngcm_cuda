
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorld(float f) 
{
	/*printf("Hello thread %d, f=%f\n", threadIdx.x, f);*/
	/* printf("Hello block %i running thread %i, f=%f\n", blockIdx.x, threadIdx.x, f);*/
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	printf("Hello block %i running thread %i, f=%f\n", blockIdx.x, idx, f);
}

int main() 
{
	dim3 grid(2, 2, 1);
	dim3 block(2, 2, 1);
	/*helloWorld<<<1, 10>>>(1.2345f);*/ 
	/*helloWorld<<<2, 5>>>(1.2345f);*/ 
	helloWorld<<<grid, block>>>(1.2345f); 
	hipDeviceReset();
	return 0;
}