
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorld(float f) 
{
	/*printf("Hello thread %d, f=%f\n", threadIdx.x, f);*/
	printf("Hello block %i running thread %i, f=%f\n", blockIdx.x, threadIdx.x, f);
}

int main() 
{
	helloWorld<<<1, 10>>>(1.2345f); 
	hipDeviceReset();
	return 0;
}